
#include <hip/hip_runtime.h>
#ifdef __CUDA_ARCH__
#undef __CUDA_ARCH__
#endif
#define __CUDA_ARCH__ 130
#ifndef CUDA_2
#define BOUNDS(a,b,c,d) __launch_bounds__( (a)*(b)*(c), d )
#else 
#define BOUNDS(a,b,c,d) 
#endif 
#define __global   
#define __private  
#define __local    
#define __constant 
#ifndef M_PI 
#define M_PI 3.14159265358979323846
#endif  
typedef unsigned int uint; 
extern "C" {
__global__ void swan_fast_fill( uint4 *ptr, int len ) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if( idx<len) {
		ptr[idx] = make_uint4( 0,0,0,0 );
	}
}

__global__ void swan_fast_fill_word( uint *ptr, int len ) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if( idx<len) {
		ptr[idx] = 0;
	}
}




__global__ void canary( int N ) {
//	int idx = threadIdx.x + blockDim.x * blockIdx.x;
//	if( idx < N ) {
//		out[idx] = in[idx];
//	}
}


}
